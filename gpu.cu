
#include<bits/stdc++.h>
#include <hip/hip_runtime.h>
#include<iostream>
#include <fstream>
#include<stdio.h>
#include<stdlib.h>

//#define f first
//#define s second

using namespace std;


struct eltype{
    int f;
    int rowno;
    int colno;
};

vector< vector <  struct eltype > >  g;

int find_col(vector< vector <  struct eltype > > A , int rows , int cols){
    int minind = -1;
    int min = 1000;
    int sum = 0;
    for(int ii=0;ii<cols;ii++){
        sum = 0;
        for(int jj=0;jj<rows;jj++){
            if(A[jj][ii].f == 1 ){
                sum = sum + 1;
            }
        }
        if(sum<min){
            min = sum;
            minind = ii;
        }
    }
    if(min==0)return -1;
    return minind;
}

vector< vector <  struct eltype > > delcol( vector< vector <  struct eltype > > v , int j ){
    // delete j indexed coloumn
    for(int ii=0;ii<v.size();ii++){
        v[ii].erase(v[ii].begin()+j);
    }
    return v;
}

vector< vector <  struct eltype > > help_algox(vector< vector <  struct eltype > > A , int rows , int cols  ,  int r){
    vector<int> x;
    //cout<<"cols selected"<<endl;
    for(int ii=0;ii<A[0].size();ii++){
        if(A[r][ii].f == 1){
            x.push_back(ii);
            //cout<<ii<<" ";
        }
    }
    //cout<<endl;
    for(int ii=x.size()-1;ii>=0;ii--){
        for(int kk=A.size()-1;kk>=0;kk--){
            if(A[kk][x[ii]].f == 1){
                // delete kk row
                A.erase(A.begin()+kk);
                //cout<<" row "<<kk<<" deleted"<<endl;
                //print2(A);
            }
        }
        // delete coloumn ii

        A = delcol(A,x[ii]);

        //cout<<"col "<<x[ii]<<" deleted"<<endl;
        //print2(A);
    }
    //vector< vector <  pair< int , pair <int , int> > > > soln;

    return A;

    //if(A.size() == 0) return partsoln;
    //return algX(A,A.size(),A[0].size(),partsoln);
}


vector< vector <  struct eltype > > algX ( vector< vector < struct eltype > > A,int rows ,int cols ,vector< vector <  struct eltype > > partsoln){
    if(cols==0){
        //cout<<" ending"<<endl;
        return partsoln;
    }

    vector< vector <  struct eltype > > soln2;

    //cout<<"partsoln size is : "<<partsoln.size()<<endl;

    //cout<<"in algx  cols are : "<<cols <<endl;
    
    // choose the coloumn with min no of 1's in it

    int c = find_col(A,rows,cols); // c is our chosen coloumn index

    if(c==-1) return soln2;

    //cout<<"selected colomn is :"<<c<<endl;

    vector<int> r;

    for(int ii=0;ii<rows;ii++){
        if(A[ii][c].f == 1){
            r.push_back(ii);
        }
    }

    vector< vector <  struct eltype > > temp1; // for copy of A
    vector< vector <  struct eltype > > temp2 ; // for copy of partsoln
    vector< vector <  struct eltype > > soln;
    vector< vector <  struct eltype > > soln1;

    //cout<<"check"<<endl;
    //cout<<r.size()<<endl;

    int useg,llpr,ttpr,rwno;

    for(int jj=0;jj<r.size();jj++){
        // call each branch
        //cout<<"each branch"<<endl;
        //cout<<"slected row is :"<<r[jj]<<endl;
        temp1 = A;
        temp2 = partsoln;
        //temp2.push_back(A[r[jj]]);
        useg = ((A[r[jj]][0]).rowno) ;
        llpr = useg/100;
        ttpr = useg%100;
        rwno = (ttpr-1)*9 + llpr-1;

        temp2.push_back(g[rwno]);
        soln = help_algox(temp1,rows,cols,r[jj]);
        //cout<<"hello"<<endl;
        //print2(soln);
        if(soln.size() == 0) { 
            //cout<<"part soln is "<<endl;
            //print2(partsoln); 
            return temp2;
        }

        soln1 =  algX(soln,soln.size() , soln[0].size() , temp2 );

        if(soln1.size() != 0) return soln1;
    }

    return soln2;

}

__global__ void kernel1(struct eltype * gpug){
    int yy = blockIdx.x;
    int ii = yy/9;
    int jj = yy%9+1;
    int kk = threadIdx.x + 1;
    gpug[ ( 9*ii+(jj-1) ) * 324 + kk - 1 ].f = 0;
    gpug[ ( 9*ii+(jj-1) ) * 324 + kk - 1 ].rowno = 1+ii+100*jj;
    gpug[ ( 9*ii+(jj-1) ) * 324 + kk - 1 ].colno = kk;
}

__global__ void kernel2(struct eltype * gpug){
    int ii = blockIdx.x;
    int jj = threadIdx.x+1;
    gpug[(9*ii+(jj-1))*324 + ii].f =1; 
    gpug[ (9*ii+(jj-1))*324 + 80+jj+(ii/9)*9  ].f = 1;
    gpug[ (9*ii+(jj-1))*324 + 161+jj+(ii%9)*9  ].f = 1;
    gpug[ (9*ii+(jj-1))*324 +  242+jj+(ii/27)*27 + ((ii/3)%3)*9 ].f = 1;
}

__global__ void kernel3(int * a ){
    int id = threadIdx.x;
    a[id]=0;
}

__global__ void kernel4(struct eltype * gpug , int * a , int * b){
    // m is ecover gpu copy which is g
    // a is gpurow
    // b is gpucol
    int ii = blockIdx.x;
    int jj = threadIdx.x;
    if(b[jj] ==1 && gpug[ii*324+jj].f == 1){
        a[ii]=1;
    }
}




int main(){
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start,0);

    struct eltype * ecover ;
    ecover = (struct eltype * )malloc (729 * 324 * sizeof(struct eltype)) ;

    struct eltype * gpug;
    hipMalloc(&gpug , 729 * 324 * sizeof(struct eltype));

    //struct eltype * gpuexactcover;
    //cudaMalloc(&gpuexactcover , 729 * 324 * sizeof(struct eltype));

    

    

    kernel1<<< 729 , 324 >>> (gpug) ;
    hipDeviceSynchronize();
    hipMemcpy(ecover , gpug , 729 * 324 * sizeof(struct eltype) , hipMemcpyDeviceToHost);


    
    // this can be done by one kernel2 81 X 9 launch
    

    kernel2<<< 81 , 9 >>> (gpug);
    hipDeviceSynchronize();
    hipMemcpy(ecover , gpug , 729 * 324 * sizeof(struct eltype) , hipMemcpyDeviceToHost);
    //int ch=0;
    //for(int ii=0;ii<729;ii++){
        //ch=0;
        //for(int jj=0;jj<324;jj++){
            //if(ecover[ii*324+jj].f == 1)ch++;
        //}
        //cout<<ch<<" ";
    //}
    //cout<<endl;
    // copy ecover to g
    // gpug is used to push a row into partsoln

    //cout<<"kernel2 done "<<endl;
    
    //cudaMemcpy( gpug , exactcover , 729 * 324 * sizeof(struct eltype) , cudaMemcpyHostToDevice )
    // gpug is used to push a row into partsoln

    int * sudoku ;
    sudoku = (int *) malloc (9 * 9 * sizeof(int));

    ifstream fin;
    fin.open("input.txt");
    int cell;

    int pos[82];int * gpupos;
    hipMalloc(&gpupos,82*sizeof(int));
    kernel3 <<< 1,82 >>>(gpupos);
    hipDeviceSynchronize();
    hipMemcpy(pos,gpupos , 82*sizeof(int) ,hipMemcpyDeviceToHost );
    
    int row[729];int * gpurow;
    hipMalloc(&gpurow,729*sizeof(int));
    kernel3 <<< 1,729 >>>(gpurow);
    hipDeviceSynchronize();
    hipMemcpy(row,gpurow , 729*sizeof(int) ,hipMemcpyDeviceToHost );

    int col[324];int * gpucol;
    hipMalloc(&gpucol,324*sizeof(int));
    kernel3 <<< 1,324 >>>(gpucol);
    hipDeviceSynchronize();
    hipMemcpy(col,gpucol , 324*sizeof(int) ,hipMemcpyDeviceToHost );

    //for(int ii=0;ii<729;ii++){
        //cout<<row[ii]<<" ";
    //}
    //cout<<endl;
    //for(int ii=0;ii<324;ii++){
        //col[ii]=0;
    //} 
    //for(int ii=0;ii<82;ii++){
        //pos[ii]=0;
   // }

    vector<int> delrows; // vector containing indexes of rows to delete
    vector<int> delcols; // vector containing indexes of cols to delete
    for(int ii=0;ii<9;ii++){
        for(int jj=0;jj<9;jj++){
            fin>>cell;
            sudoku[ii*9 + jj] = cell;
            if(cell!=0){
                pos[ii*9 + jj + 1]=1;
                //row[(ii*9 + jj )*9 ]=1;
                //for(int kk=0;kk<9;kk++){
                    //row[(ii*9 + jj )*9 + kk ]=1;
                    //delrows.push_back((ii*9 + jj )*9 + kk);
                //}
                // ii above is ii*9+jj
                // jj above is cell
                col[ii*9 + jj]=1;
                col[80+cell+((ii*9+jj)/9)*9]=1;
                col[161+cell+((ii*9+jj)%9)*9]=1;
                col[242+cell+((ii*9+jj)/27)*27 + (((ii*9+jj)/3)%3)*9]=1;
                //delrows.push_back()
            }
        }
    }

    

    hipMemcpy(gpucol , col , 324*sizeof(int) , hipMemcpyHostToDevice);
    hipMemcpy(gpurow , row , 729*sizeof(int) , hipMemcpyHostToDevice);
    hipMemcpy(gpug , ecover , 729*324*sizeof(struct eltype) , hipMemcpyHostToDevice);
    

    kernel4<<<729 , 324 >>> (gpug , gpurow , gpucol);
    hipDeviceSynchronize();
    hipMemcpy(row,gpurow , 729*sizeof(int) ,hipMemcpyDeviceToHost );

    

    for(int ii=0;ii<324;ii++){
        //cout<<col[ii]<<" ";
        if(col[ii]==1){
            delcols.push_back(ii);
        }
    }
    //cout<<endl;
    

    vector <  struct eltype > temp45;

    vector< vector <  struct eltype > > exactcover ; // exact cover matrix of input sudoku

    int check =0;
    for(int ii=0;ii<729;ii++){
        exactcover.push_back(temp45);
        
    }
    for(int ii=0;ii<729;ii++){
        //check=0;
        for(int jj=0;jj<324;jj++){
            exactcover[ii].push_back(ecover[ii*324 + jj]);
            //if( jj <81 && exactcover[ii][jj].f==1)check++;
        }
        //cout<<check<<" ";
    }
    //cout<<endl;
    g = exactcover;
    cout<<exactcover[0].size()<<endl;

    
    

    for(int ii=0;ii<729;ii++){
        if(row[ii]==1){
            delrows.push_back(ii);
        }
    }

    cout<<delrows.size()<<" "<<delcols.size()<<endl;

    for(int ii=delrows.size()-1;ii>=0 ;ii--){
        exactcover.erase(exactcover.begin() + delrows[ii]);
    }

    for(int ii=delcols.size()-1;ii>=0;ii--){
        exactcover = delcol(exactcover , delcols[ii]);
    }
    cout<<exactcover.size()<<" ";
    cout<<exactcover[0].size()<<endl;

    



    vector< vector <  struct eltype > > empty;
    vector< vector <  struct eltype > > soln = algX(exactcover , exactcover.size() , exactcover[0].size() , empty);

    cout<<"soln is"<<endl;
    cout<<soln.size()<<endl;

    

    // one kernel can write into sudoku using soln
    int sdk,bdk,cdk,edk,fdk;
    for(int ii=0;ii<soln.size();ii++){
        for(int jj=0;jj<81;jj++){
            if(soln[ii][jj].f == 1){
                //cout<<ii<<" "<<jj<<endl;
                // add an element 
                sdk = (soln[ii][jj].rowno) ; // row no as stored in exact cover matrix
                sdk = sdk-1;
                bdk = sdk/100; // the no to put
                cdk = sdk%100;
                edk = cdk/9; // row index of sudoku
                fdk = cdk%9; // col index of sudoku
                sudoku[edk*9+fdk]=bdk;
                //cout<<edk<<" "<<fdk<<" "<<bdk<<endl;
            }
        }
    }

    for(int ii=0;ii<9;ii++){
        for(int jj=0;jj<9;jj++){
            cout<<sudoku[ii*9+jj]<<" ";
        }
        cout<<endl;
    }
    //print2d(sudoku);

    fin.close();

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken by function to execute is: %.6f ms\n", milliseconds);
    return 0;
}