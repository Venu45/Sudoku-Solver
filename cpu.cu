
// sudoku solver sequential execution , using algorithm x , exact cover

#include<bits/stdc++.h>
#include<hip/hip_runtime.h>
#include <algorithm>
#include <chrono>
#include<iostream>
#include <fstream>
#include<stdio.h>
#include<stdlib.h>

#define f first
#define s second

using namespace std;
//using namespace std::chrono;

vector< vector <  pair< int , pair <int , int> > > >  g;

void print1d(vector<int> v){
    for(int ii=0;ii<v.size();ii++){
        cout<<v[ii]<<" ";
    }
    cout<<endl;
}

void print2d(vector < vector < int> >v){
    for(int ii=0;ii<v.size();ii++){
        for(int jj=0;jj<v[ii].size();jj++){
            cout<<v[ii][jj]<<" ";
        }
        cout<<endl;
    }
}

void print2(vector< vector <  pair< int , pair <int , int> > > > v){
    if(v.size()==0)return;
    cout<<"   ";
    for(int jj=0;jj<v[0].size();jj++){
        cout<<(v[0][jj].s).s<<"  ";
    }
    cout<<endl;
    for(int ii=0;ii<v.size();ii++){
        cout<<(v[ii][0].s).f<<"  ";
        for(int jj=0;jj<v[ii].size();jj++){
            cout<<v[ii][jj].f<<"  ";
        }
        cout<<endl;
    }
    cout<<endl;
}

int find_col(vector< vector <  pair< int , pair <int , int> > > > A , int rows , int cols){
    int minind = -1;
    int min = 1000;
    int sum = 0;
    for(int ii=0;ii<cols;ii++){
        sum = 0;
        for(int jj=0;jj<rows;jj++){
            if(A[jj][ii].f == 1 ){
                sum = sum + 1;
            }
        }
        if(sum<min){
            min = sum;
            minind = ii;
        }
    }
    if(min==0)return -1;
    return minind;
}

vector< vector <  pair< int , pair <int , int> > > > delcol( vector< vector <  pair< int , pair <int , int> > > > v , int j ){
    // delete j indexed coloumn
    for(int ii=0;ii<v.size();ii++){
        v[ii].erase(v[ii].begin()+j);
    }
    return v;
}

/*
If the matrix A has no columns, the current partial solution is a valid solution; terminate successfully.
Otherwise choose a column c (deterministically).
Choose a row r such that Ar, c = 1 (nondeterministically).
Include row r in the partial solution.
For each column j such that Ar, j = 1,
    for each row i such that Ai, j = 1,
        delete row i from matrix A.
    delete column j from matrix A.
Repeat this algorithm recursively on the reduced matrix A.
 */

vector< vector <  pair< int , pair <int , int> > > > help_algox(vector< vector <  pair< int , pair <int , int> > > > A , int rows , int cols  , vector< vector <  pair< int , pair <int , int> > > > partsoln , int r){
    vector<int> x;
    //cout<<"cols selected"<<endl;
    for(int ii=0;ii<A[0].size();ii++){
        if(A[r][ii].f == 1){
            x.push_back(ii);
            //cout<<ii<<" ";
        }
    }
    //cout<<endl;
    for(int ii=x.size()-1;ii>=0;ii--){
        for(int kk=A.size()-1;kk>=0;kk--){
            if(A[kk][x[ii]].f == 1){
                // delete kk row
                A.erase(A.begin()+kk);
                //cout<<" row "<<kk<<" deleted"<<endl;
                //print2(A);
            }
        }
        // delete coloumn ii
        A = delcol(A,x[ii]);
        //cout<<"col "<<x[ii]<<" deleted"<<endl;
        //print2(A);
    }
    //vector< vector <  pair< int , pair <int , int> > > > soln;
    return A;
    //if(A.size() == 0) return partsoln;
    //return algX(A,A.size(),A[0].size(),partsoln);
}


vector< vector <  pair< int , pair <int , int> > > > algX ( vector< vector <  pair< int , pair <int , int> > > > A , int rows , int cols  , vector< vector <  pair< int , pair <int , int> > > > partsoln ){
    if(cols==0){
        //cout<<" ending"<<endl;
        return partsoln;
    }

    vector< vector <  pair< int , pair <int , int> > > > soln2;

    //cout<<"partsoln size is : "<<partsoln.size()<<endl;

    //cout<<"in algx  cols are : "<<cols <<endl;
    
    // choose the coloumn with min no of 1's in it

    int c = find_col(A,rows,cols); // c is our chosen coloumn index

    if(c==-1) return soln2;

    //cout<<"selected colomn is :"<<c<<endl;

    vector<int> r;

    for(int ii=0;ii<rows;ii++){
        if(A[ii][c].f == 1){
            r.push_back(ii);
        }
    }

    vector< vector <  pair< int , pair <int , int> > > > temp1; // for copy of A
    vector< vector <  pair< int , pair <int , int> > > > temp2 ; // for copy of partsoln
    vector< vector <  pair< int , pair <int , int> > > > soln;
    vector< vector <  pair< int , pair <int , int> > > > soln1;

    //cout<<"check"<<endl;
    //cout<<r.size()<<endl;

    int useg,llpr,ttpr,rwno;

    for(int jj=0;jj<r.size();jj++){
        // call each branch
        //cout<<"each branch"<<endl;
        //cout<<"slected row is :"<<r[jj]<<endl;
        temp1 = A;
        temp2 = partsoln;
        //temp2.push_back(A[r[jj]]);
        useg = ((A[r[jj]][0]).s).f ;
        llpr = useg/100;
        ttpr = useg%100;
        rwno = (ttpr-1)*9 + llpr-1;

        temp2.push_back(g[rwno]);
        soln = help_algox(temp1,rows,cols,temp2,r[jj]);
        //cout<<"hello"<<endl;
        //print2(soln);
        if(soln.size() == 0) { 
            //cout<<"part soln is "<<endl;
            //print2(partsoln); 
            return temp2;
        }
        soln1 =  algX(soln,soln.size() , soln[0].size() , temp2 );
        if(soln1.size() != 0) return soln1;
    }

    return soln2;

}



int main(){

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start,0);
    

    //vector< vector <  pair< int , pair <int , int> > > > temp;

    //auto start = chrono::high_resolution_clock::now();

    vector <  pair< int , pair <int , int> > > temp;

    vector< vector <  pair< int , pair <int , int> > > > exactcover ; // exact cover matrix of input sudoku

    for(int ii=0;ii<729;ii++){
        exactcover.push_back(temp);
    }
    // exact cover has 729 rows
    for(int ii=0;ii<81;ii++){
        for(int jj=1;jj<=9;jj++){
            for(int kk=1;kk<=324;kk++){
                exactcover[9*ii+(jj-1)].push_back(make_pair( 0 , make_pair( 1+ii+100*jj , kk) ) );
            }
        }
    }
    //print2(exactcover);

    for(int ii=0;ii<81;ii++){
        //cout<<"row number is :"<<ii<<endl;
        for(int jj=1;jj<=9;jj++){
            // through all 729 rows keep 41's in each row
            exactcover[9*ii+(jj-1)][ii].f = 1;
            exactcover[9*ii+(jj-1)][80+jj+(ii/9)*9].f = 1;
            //cout<<80+jj+(ii/9)*9<<" ";
            exactcover[9*ii+(jj-1)][161+jj+(ii%9)*9].f = 1;
            //cout<<161+jj+(ii%9)*9<<" ";
            exactcover[9*ii+(jj-1)][242+jj+(ii/27)*27 + ((ii/3)%3)*9].f = 1;
            //cout<<"row number is :"<<ii<<endl;
            //cout<< 242+jj+(ii/27)*27 + ((ii/3)%3)*9<<" ";
        }
        //cout<<endl;
    }

    g = exactcover;
    //print2(exactcover);

    ofstream fout;
    ifstream fin;
    int cell;
    vector< vector <int> > sudoku;
    vector <int> p;
    for(int ii=0;ii<9;ii++){
        sudoku.push_back(p);
    }
    fin.open("input.txt");
    //fout.open("output.txt");
    int pos[82];
    int row[729];
    int col[324];
    for(int ii=0;ii<729;ii++){
        row[ii]=0;
    }
    for(int ii=0;ii<324;ii++){
        col[ii]=0;
    } 
    for(int ii=0;ii<82;ii++){
        pos[ii]=0;
    }
    vector<int> delrows; // vector containing indexes of rows to delete
    vector<int> delcols; // vector containing indexes of cols to delete
    for(int ii=0;ii<9;ii++){
        for(int jj=0;jj<9;jj++){
            fin>>cell;
            sudoku[ii].push_back(cell);
            if(cell!=0){
                pos[ii*9 + jj + 1]=1;
                //row[(ii*9 + jj )*9 ]=1;
                /*for(int kk=0;kk<9;kk++){
                    row[(ii*9 + jj )*9 + kk ]=1;
                    //delrows.push_back((ii*9 + jj )*9 + kk);
                }*/
                // ii above is ii*9+jj
                // jj above is cell
                col[ii*9 + jj]=1;
                col[80+cell+((ii*9+jj)/9)*9]=1;
                col[161+cell+((ii*9+jj)%9)*9]=1;
                col[242+cell+((ii*9+jj)/27)*27 + (((ii*9+jj)/3)%3)*9]=1;
                //delrows.push_back()
            }
        }
    }

    for(int ii=0;ii<324;ii++){
        if(col[ii]==1){
            delcols.push_back(ii);
        }
    }
    for(int ii=0;ii<729;ii++){
        for(int jj=0;jj<delcols.size();jj++){
            if(exactcover[ii][delcols[jj]].f==1){
                // need to delete that row
                row[ii]=1;
            }
        }
    }

    for(int ii=0;ii<729;ii++){
        if(row[ii]==1){
            delrows.push_back(ii);
        }
    }

    cout<<delrows.size()<<" "<<delcols.size()<<endl;

    for(int ii=delrows.size()-1;ii>=0 ;ii--){
        exactcover.erase(exactcover.begin() + delrows[ii]);
    }

    for(int ii=delcols.size()-1;ii>=0;ii--){
        exactcover = delcol(exactcover , delcols[ii]);
    }

    print2d(sudoku);


    cout<<exactcover.size()<<endl;
    cout<<exactcover[0].size()<<endl;

    vector< vector <  pair< int , pair <int , int> > > > empty;
    vector< vector <  pair< int , pair <int , int> > > > soln = algX(exactcover , exactcover.size() , exactcover[0].size() , empty);
    cout<<"soln is"<<endl;
    cout<<soln.size()<<endl;
    int sdk,bdk,cdk,edk,fdk;
    for(int ii=0;ii<soln.size();ii++){
        for(int jj=0;jj<81;jj++){
            if(soln[ii][jj].f == 1){
                // add an element 
                sdk = (soln[ii][jj].s).f ; // row no as stored in exact cover matrix
                sdk = sdk-1;
                bdk = sdk/100; // the no to put
                cdk = sdk%100;
                edk = cdk/9; // row index of sudoku
                fdk = cdk%9; // col index of sudoku
                sudoku[edk][fdk]=bdk;
                cout<<edk<<" "<<fdk<<" "<<bdk<<endl;
            }
        }
    }
    print2d(sudoku);
    //print2(soln);



    




    

    fin.close();

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken by function to execute is: %.6f ms\n", milliseconds);

    
    return 0;
}